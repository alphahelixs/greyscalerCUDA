#include "hip/hip_runtime.h"
#include "convert_batch.hpp"

#include <tiffio.h>
#include <filesystem>
#include <iostream>
#include <vector>
#include <string>
#include <cassert>

#include <algorithm>
#include <cctype>

namespace fs = std::filesystem;

//------------------------------------------------------------------------------
// Host-side helpers
//------------------------------------------------------------------------------

std::vector<std::string>
listTiffFiles(const std::string& directory) {
    std::vector<std::string> out;
    for (const auto& entry : fs::directory_iterator(directory)) {
        if (!entry.is_regular_file()) continue;
        auto ext = entry.path().extension().string();
        std::transform(ext.begin(), ext.end(), ext.begin(), ::tolower);
        if (ext == ".tif" || ext == ".tiff")
            out.push_back(entry.path().string());
    }
    return out;
}

//------------------------------------------------------------------------------
// Single-image pipeline
//------------------------------------------------------------------------------
void processImage(const std::string& inPath,
                  const std::string& outPath,
                  int threadsPerBlock) {
    TIFF* tif = TIFFOpen(inPath.c_str(), "r");
    if (!tif) {
        std::cerr << "ERROR: Cannot open “" << inPath << "”\n";
        return;
    }

    uint32_t w=0, h=0;
    TIFFGetField(tif, TIFFTAG_IMAGEWIDTH, &w);
    TIFFGetField(tif, TIFFTAG_IMAGELENGTH, &h);
    int total = w * h;

    uint32_t* h_raster = (uint32_t*) _TIFFmalloc(total * sizeof(uint32_t));
    uint8_t*  h_gray   = (uint8_t*)  std::malloc(total * sizeof(uint8_t));
    assert(h_raster && h_gray);

    if (!TIFFReadRGBAImageOriented(tif, w, h, h_raster,
                                   ORIENTATION_TOPLEFT, 0)) {
        std::cerr << "ERROR: TIFFReadRGBAImage failed for “"
                  << inPath << "”\n";
        TIFFClose(tif);
        return;
    }
    TIFFClose(tif);

    uint32_t* d_raster = nullptr;
    uint8_t*  d_gray   = nullptr;
    hipMalloc(&d_raster, total * sizeof(uint32_t));
    hipMalloc(&d_gray,   total * sizeof(uint8_t));

    hipMemcpy(d_raster, h_raster,
               total * sizeof(uint32_t),
               hipMemcpyHostToDevice);

    int blocks = (total + threadsPerBlock - 1) / threadsPerBlock;
    rgb2grayKernel<<<blocks, threadsPerBlock>>>(d_raster, d_gray, total);
    hipDeviceSynchronize();

    hipMemcpy(h_gray, d_gray,
               total * sizeof(uint8_t),
               hipMemcpyDeviceToHost);

    TIFF* out = TIFFOpen(outPath.c_str(), "w");
    if (out) {
        TIFFSetField(out, TIFFTAG_IMAGEWIDTH,       w);
        TIFFSetField(out, TIFFTAG_IMAGELENGTH,      h);
        TIFFSetField(out, TIFFTAG_SAMPLESPERPIXEL,  1);
        TIFFSetField(out, TIFFTAG_BITSPERSAMPLE,    8);
        TIFFSetField(out, TIFFTAG_ORIENTATION,      ORIENTATION_TOPLEFT);
        TIFFSetField(out, TIFFTAG_PHOTOMETRIC,      PHOTOMETRIC_MINISBLACK);
        TIFFSetField(out, TIFFTAG_PLANARCONFIG,     PLANARCONFIG_CONTIG);

        tsize_t linebytes = w * sizeof(uint8_t);
        uint8_t* buf = (uint8_t*) _TIFFmalloc(linebytes);
        for (uint32_t row = 0; row < h; ++row) {
            memcpy(buf, h_gray + row * w, linebytes);
            TIFFWriteScanline(out, buf, row, 0);
        }
        _TIFFfree(buf);
        TIFFClose(out);
    } else {
        std::cerr << "ERROR: Cannot open “" << outPath << "” for write\n";
    }

    _TIFFfree(h_raster);
    free(h_gray);
    hipFree(d_raster);
    hipFree(d_gray);

    std::cout << "Done: “" << inPath << "” → “" << outPath << "”\n";
}

//------------------------------------------------------------------------------
// CUDA kernel
//------------------------------------------------------------------------------
__global__
void rgb2grayKernel(const uint32_t* raster,
                    uint8_t*      gray,
                    int           totalPixels) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalPixels) return;

    uint32_t pix = raster[idx];
    float r = float((pix      ) & 0xFF);
    float g = float((pix >>  8) & 0xFF);
    float b = float((pix >> 16) & 0xFF);
    gray[idx] = uint8_t(fminf(0.3f*r + 0.59f*g + 0.11f*b, 255.0f));
}

//------------------------------------------------------------------------------
// main: batch over folder
//------------------------------------------------------------------------------
int main(int argc, char** argv) {
    if (argc < 3) {
        std::cerr << "Usage: " << argv[0]
                  << " <input_dir> <output_dir> [-t threadsPerBlock]\n";
        return 1;
    }

    std::string inDir  = argv[1];
    std::string outDir = argv[2];
    int threadsPerBlock = 256;
    for (int i = 3; i+1 < argc; i += 2) {
        if (std::string(argv[i]) == "-t")
            threadsPerBlock = std::stoi(argv[i+1]);
    }

    fs::create_directories(outDir);
    auto files = listTiffFiles(inDir);
    for (auto& inPath : files) {
        auto stem = fs::path(inPath).stem().string();
        std::string outPath = outDir + "/" + stem + "_gray.tiff";
        processImage(inPath, outPath, threadsPerBlock);
    }
    return 0;
}
